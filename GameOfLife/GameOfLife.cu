#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <SFML/Graphics.hpp>
#include <stdio.h>
#include <time.h>
#include <random>
#include <unordered_map>
#include "GameOfLife.h"
#define window_area state.window_w * state.window_h
#define pixels_size 4 * (window_area)

struct GOL_params {
	const int WINDOW_WIDTH = 512;
	const int WINDOW_HEIGHT = 512;
	const int BOARD_SIZE = 8192;
	const int BOARD_AREA = BOARD_SIZE * BOARD_SIZE;
	const std::string WINDOW_TITLE = "Geim of Life";
	const int TARGET_FPS = 120;
	const double ZOOM_FACTOR = 1.25;

	uint8_t* pixels;
	uint8_t* board;
	uint8_t* dev_board_IN;
	uint8_t* dev_board_OUT;
	uint8_t* dev_pixels;

	sf::RenderWindow window;
	sf::ContextSettings settings;
	sf::Sprite mainSprite;
	sf::Texture mainTexture;
	sf::Vector2i MouseMoveStartPos;

	int window_w = WINDOW_WIDTH;
	int window_h = WINDOW_HEIGHT;

	double simulation_fps = 5.0;

	bool paused = false;
	bool LMousePressed = false;
	bool MMousePressed = false;
	bool resetRect = false;
	bool upload_and_redraw_board = false;
	bool redraw_board = false;
	bool fullScreen = false;
	bool showSettings = false;

	int drawMode = -1;
	int lastBoardIndx = -1;

	double xs = 0.0;
	double ys = 0.0;
	double dx = 1.0;
	double dy = dx * window_h / window_w;

	clock_t last_frame_time = clock();
	clock_t cur_time = clock();

	sf::Font font;
	sf::RectangleShape settings_button;
	sf::Text settings_button_text;

	std::uniform_int_distribution<> distributed_random;
	std::mt19937 random_generator_function;

	const std::unordered_map<sf::Keyboard::Key, int> keyToPatternMap = {
		{sf::Keyboard::R, 0},
		{sf::Keyboard::Num0, 0},
		{sf::Keyboard::Num1, 1},
		{sf::Keyboard::Num2, 2},
		{sf::Keyboard::Num3, 3},
		{sf::Keyboard::Num4, 4}
	};
};

__global__ void get_next_frame_kernel(uint8_t* board_1, uint8_t* board_2, int BOARD_SIZE) {
	int neigh = 0;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int cx = x;
	int cy = y;

	for (int dy = -1; dy <= 1; dy++) {
		cy = y + dy;
		for (int dx = -1; dx <= 1; dx++) {
			cx = x + dx;
			if (cx >= 0 && cx <= BOARD_SIZE - 1 && cy >= 0 && cy <= BOARD_SIZE - 1 && (dx != 0 || dy != 0)) {
				neigh += board_1[cy * BOARD_SIZE + cx] != 0;
			}
		}
	}

	int tid = BOARD_SIZE * y + x;
	uint8_t cellStatus = board_1[tid];
	if (cellStatus == 0 && neigh == 3) {
		board_2[tid] = 1;
	}
	else if (cellStatus == 1 && (neigh < 2 || neigh > 3)) {
		board_2[tid] = 0;
	}
	else {
		board_2[tid] = cellStatus;
	}
}
__global__ void update_pixels_kernel(uint8_t* pixels, uint8_t* board, int BOARD_SIZE, int w_width, int w_height, double xs, double ys, double dx, double dy) {
	int wx = blockIdx.x * blockDim.x + threadIdx.x;
	int wy = blockIdx.y * blockDim.y + threadIdx.y;

	int pixel_index = 4 * (wy * w_width + wx);

	if (wx >= w_width || wy >= w_height) {
		return;
	}

	uint8_t red;
	uint8_t blue;
	uint8_t green;

	double ww = w_width;
	double wh = w_height;

	double xb = xs + (double)wx / ww * dx; // [0 : 1]
	double yb = ys + (double)wy / wh * dy; // [0 : 1]

	if (xb < 0.0 || yb < 0.0 || xb >= 1.0 || yb >= 1.0) {
		pixels[pixel_index] = 0;
		pixels[pixel_index + 1] = 0;
		pixels[pixel_index + 2] = 0;
		pixels[pixel_index + 3] = 255;
		return;
	}

	int board_index_x = xb * BOARD_SIZE;
	int board_index_y = yb * BOARD_SIZE;
	int board_index = board_index_y * BOARD_SIZE + board_index_x;

	uint8_t status = board[board_index];

	switch (status) {
	case 0:
		red = 250;
		green = 250;
		blue = 250;
		break;
	case 1:
		red = 0;
		green = 0;
		blue = 0;
		break;
	default:
		red = status;
		green = status;
		blue = status;
		break;
	}

	pixels[pixel_index] = red;
	pixels[pixel_index + 1] = green;
	pixels[pixel_index + 2] = blue;
	pixels[pixel_index + 3] = 255;
}

hipError_t get_next_frame_cuda(GOL_params& state) {
	hipError_t cudaStatus;

	dim3 blockSize(16, 16);
	dim3 gridSize((state.BOARD_SIZE + blockSize.x - 1) / blockSize.x, (state.BOARD_SIZE + blockSize.y - 1) / blockSize.y);

	get_next_frame_kernel << <gridSize, blockSize >> > (state.dev_board_IN, state.dev_board_OUT, state.BOARD_SIZE);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("error in hipGetLastError\n");
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(state.board, state.dev_board_OUT, state.BOARD_AREA, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("error in hipMemcpy\n");
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("error in hipDeviceSynchronize\n");
		return cudaStatus;
	}

	uint8_t* cuda_board_IN_temp = state.dev_board_IN;
	state.dev_board_IN = state.dev_board_OUT;
	state.dev_board_OUT = cuda_board_IN_temp;

	return cudaStatus;
}
hipError_t update_pixels_cuda(GOL_params& state) {
	hipError_t cudaStatus;

	dim3 blockSize(16, 16);
	dim3 gridSize((state.window_w + blockSize.x - 1) / blockSize.x, (state.window_h + blockSize.y - 1) / blockSize.y);

	update_pixels_kernel << <gridSize, blockSize >> > (state.dev_pixels, state.dev_board_IN, state.BOARD_SIZE, state.window_w, state.window_h, state.xs, state.ys, state.dx, state.dy);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("error in update_pixels_kernel\n");
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("error in hipDeviceSynchronize update_pixels_cuda\n");
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(state.pixels, state.dev_pixels, pixels_size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("error in hipMemcpy state.pixels\n");
		return cudaStatus;
	}

	return cudaStatus;
}
void resize_window(GOL_params& state, int w, int h) {
	state.window_w = w;
	state.window_h = h;
	state.window.setView(sf::View(sf::FloatRect(0, 0, (float)state.window_w, (float)state.window_h)));
}
void make_window(GOL_params& state) {
	state.window.close();
	sf::VideoMode screenSize;
	if (state.fullScreen) {
		screenSize = sf::VideoMode::getDesktopMode();
		state.window.create(screenSize, state.WINDOW_TITLE, sf::Style::Fullscreen, state.settings);
	}
	else {
		screenSize = sf::VideoMode(state.window_w, state.window_h, 24);
		state.window.create(screenSize, state.WINDOW_TITLE, sf::Style::Resize | sf::Style::Close, state.settings);
	}
	resize_window(state, screenSize.width, screenSize.height);
	state.window.setFramerateLimit(state.TARGET_FPS);
	state.window.setKeyRepeatEnabled(false);
	state.window.requestFocus();
}
void sfml_setup_gui(GOL_params& state) {
	state.settings_button = sf::RectangleShape(sf::Vector2f(60, 25));
	if (!state.font.loadFromFile("arial.ttf")) {
		printf("Font file not found!");
	}
	state.settings_button_text = sf::Text("settings", state.font, 15);
	state.settings_button.setFillColor(sf::Color(128, 128, 128, 128));
	state.settings_button.setPosition(0, 0);
}
hipError_t cuda_setup(GOL_params& state) {
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&state.dev_board_IN, state.BOARD_AREA);
	if (cudaStatus != hipSuccess) {
		printf("error in hipMalloc state.dev_board_IN cuda_setup\n");
	}

	cudaStatus = hipMalloc((void**)&state.dev_board_OUT, state.BOARD_AREA);
	if (cudaStatus != hipSuccess) {
		printf("error in hipMalloc state.dev_board_OUT cuda_setup\n");
	}

	cudaStatus = hipMalloc((void**)&state.dev_pixels, pixels_size);
	if (cudaStatus != hipSuccess) {
		printf("error in hipMalloc state.dev_pixels cuda_setup\n");
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("error in hipDeviceSynchronize cuda_setup\n");
	}

	return cudaStatus;
}
void sfml_setup(GOL_params& state) {
	state.settings.depthBits = 24;
	state.settings.stencilBits = 8;
	state.settings.antialiasingLevel = 4;
	state.settings.majorVersion = 3;
	state.settings.minorVersion = 0;

	make_window(state); 
	sfml_setup_gui(state);
}
void setup(GOL_params& state) {
	std::random_device rd;
	state.random_generator_function = std::mt19937(rd());
	state.distributed_random = std::uniform_int_distribution<>(0, 1);

	state.board = (uint8_t*)malloc(state.BOARD_AREA);
	if (state.board == NULL) {
		printf("error in malloc state.board\n");
	}
	state.pixels = (uint8_t*)malloc(pixels_size);
	if (state.pixels == NULL) {
		printf("error in malloc state.pixels\n");
	}

	sfml_setup(state);
	cuda_setup(state);
}
hipError_t cuda_resize(GOL_params& state) {
	hipError_t cudaStatus;
	hipFree(state.dev_pixels);

	cudaStatus = hipMalloc((void**)&state.dev_pixels, pixels_size);
	if (cudaStatus != hipSuccess) {
		printf("error in hipMalloc state.dev_pixels cuda_resize\n");
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("error in hipDeviceSynchronize cuda_resize\n");
	}
	return cudaStatus;
}
int mouse_cords_to_board_indx(GOL_params& state, sf::Vector2i& mousePos) {
	double xm = (double)mousePos.x / state.window_w;
	double ym = (double)mousePos.y / state.window_h;
	int board_x = (state.xs + xm * state.dx) * state.BOARD_SIZE;
	int board_y = (state.ys + ym * state.dy) * state.BOARD_SIZE;
	int board_indx = board_y * state.BOARD_SIZE + board_x;

	return board_indx;
}
void board_reset_and_draw_pattern(GOL_params& state, int indx) {
	memset(state.board, 0, state.BOARD_AREA);
	switch (indx)
	{
	case 0:
		break;
	case 1:
		state.board[state.BOARD_SIZE * 100 + 100] = 1;
		state.board[state.BOARD_SIZE * 100 + 101] = 1;
		state.board[state.BOARD_SIZE * 100 + 102] = 1;
		break;
	case 2:
		for (int iy = state.BOARD_SIZE / 3; iy < 2 * state.BOARD_SIZE / 3; iy++) {
			for (int ix = state.BOARD_SIZE / 3; ix < 2 * state.BOARD_SIZE / 3; ix++) {
				state.board[iy * state.BOARD_SIZE + ix] = state.distributed_random(state.random_generator_function);
			}
		}
		break;
	case 3:
		for (int ix = state.BOARD_SIZE / 3; ix < 2 * state.BOARD_SIZE / 3; ix++) {
			state.board[state.BOARD_SIZE * state.BOARD_SIZE / 2 + ix] = 1;
		}
		break;
	case 4:
		for (int iy = 0; iy < state.BOARD_SIZE; iy++) {
			state.board[iy * state.BOARD_SIZE + state.BOARD_SIZE / 2] = 1;
		}
		break;
	}
	state.upload_and_redraw_board = true;
}
void handle_board_reset_pattern(GOL_params& state, sf::Keyboard::Key key) {
	auto it = state.keyToPatternMap.find(key);
	if (it != state.keyToPatternMap.end()) {
		board_reset_and_draw_pattern(state, it->second);
	}
}
void handle_window_resize(GOL_params& state, int width, int height, bool recreate_window) {
	if (recreate_window) {
		state.window_w = width;
		state.window_h = height;
		make_window(state);
	}
	else {
		resize_window(state, width, height);
	}
	cuda_resize(state);

	state.dy = state.dx * state.window_h / state.window_w;

	state.pixels = (uint8_t*)realloc(state.pixels, pixels_size);
	state.mainTexture.create(state.window_w, state.window_h);
	state.resetRect = true;
}
void toggle_fullscreen(GOL_params& state) {
	state.fullScreen = !state.fullScreen;
	handle_window_resize(state, state.WINDOW_HEIGHT, state.WINDOW_WIDTH, true);
}
void handle_mouse_press(GOL_params& state, sf::Mouse::Button mouse_button) {
	switch (mouse_button) {
	case sf::Mouse::Left: {
		sf::Vector2i mousePos = sf::Mouse::getPosition(state.window);

		if (state.settings_button.getGlobalBounds().contains((sf::Vector2f)mousePos)) {
			printf("Button clicked!\n");
			state.showSettings = !state.showSettings;
			break;
		}

		state.LMousePressed = true;

		int board_indx = mouse_cords_to_board_indx(state, mousePos);
		state.lastBoardIndx = board_indx;

		state.drawMode = (int)(state.board[board_indx] == 0);

		state.board[board_indx] = state.drawMode;
		state.upload_and_redraw_board = true;

		break;
	}
	case sf::Mouse::Middle:
		state.MMousePressed = true;
		state.MouseMoveStartPos = sf::Mouse::getPosition(state.window);
		break;
	}
}
void handle_mouse_release(GOL_params& state, sf::Mouse::Button mouse_button) {
	switch (mouse_button) {
	case sf::Mouse::Left:
		state.LMousePressed = false;
		state.upload_and_redraw_board = true;
		state.drawMode = -1;
		break;
	case sf::Mouse::Middle:
		state.MMousePressed = false;
		state.redraw_board = true;
		break;
	}
}
void handle_mouse_move(GOL_params& state) {
	if (state.MMousePressed) {
		sf::Vector2i mousePos = sf::Mouse::getPosition(state.window);
		state.xs += (double)(state.MouseMoveStartPos.x - mousePos.x) / state.window_w * state.dx;
		state.ys += (double)(state.MouseMoveStartPos.y - mousePos.y) / state.window_h * state.dy;
		state.MouseMoveStartPos = mousePos;
		state.redraw_board = true;
	}
	if (state.LMousePressed) {
		sf::Vector2i mousePos = sf::Mouse::getPosition(state.window);
		int board_indx = mouse_cords_to_board_indx(state, mousePos);
		if (board_indx != state.lastBoardIndx) {
			state.lastBoardIndx = board_indx;
			state.board[board_indx] = state.drawMode;
			state.upload_and_redraw_board = true;
		}
	}
}
void handle_mouse_scroll(GOL_params& state, int scroll) {
	sf::Vector2i mousePos = sf::Mouse::getPosition(state.window);
	double xm = (double)mousePos.x / state.window_w;
	double ym = (double)mousePos.y / state.window_h;

	if (scroll < 0) { // zoom out
		state.xs = state.xs - xm * state.dx * (state.ZOOM_FACTOR - 1.0);
		state.ys = state.ys - ym * state.dy * (state.ZOOM_FACTOR - 1.0);

		state.dx *= state.ZOOM_FACTOR;
		state.dy *= state.ZOOM_FACTOR;
	}
	else if (scroll > 0) { // zoom in
		state.dx /= state.ZOOM_FACTOR;
		state.dy /= state.ZOOM_FACTOR;

		state.xs = state.xs + xm * state.dx * (state.ZOOM_FACTOR - 1.0);
		state.ys = state.ys + ym * state.dy * (state.ZOOM_FACTOR - 1.0);
	}
	printf("dx / dy : %lf\n", state.dx / state.dy);
	state.upload_and_redraw_board = true;
}
void handle_key_release(GOL_params& state, sf::Keyboard::Key key) {
	switch (key)
	{
	case sf::Keyboard::Space:
		state.paused = !state.paused;
		break;
	case sf::Keyboard::O:
		state.xs = 0.0;
		state.ys = 0.0;
		state.dx = 1.0;
		state.dy = state.dx * state.window_h / state.window_w;
		state.upload_and_redraw_board = true;
		break;
	case sf::Keyboard::Left:
		state.simulation_fps /= 1.5;
		printf("simulation Rate: %lf\n", state.simulation_fps);
		break;
	case sf::Keyboard::Right:
		state.simulation_fps *= 1.5;
		printf("simulation Rate: %lf\n", state.simulation_fps);
		break;
	case sf::Keyboard::R:
	case sf::Keyboard::Num0:
	case sf::Keyboard::Num1:
	case sf::Keyboard::Num2:
	case sf::Keyboard::Num3:
	case sf::Keyboard::Num4:
		handle_board_reset_pattern(state, key);
		break;
	case sf::Keyboard::F11:
		toggle_fullscreen(state);
		break;
	}
}
void draw_settings_sprite(GOL_params& state) {
	if (state.showSettings) {
		const char* settings_options[] = {
			"Board Size:",
			"Start Pattern 1",
			"Start Pattern 2",
			"Start Pattern 3",
			"Start Pattern 4",
			"Simulation Speed:"
		};

		sf::FloatRect settings_button_coords = state.settings_button.getGlobalBounds();
		float y_cord = settings_button_coords.top + settings_button_coords.height;
		float start_y_cord = y_cord;

		const int text_y_size = sizeof(settings_options) / sizeof(settings_options[0]);
		float text_width = 0.0f;

		sf::Text option_texts[text_y_size];

		for (int i = 0; i < text_y_size; ++i) {
			sf::Text option_text = sf::Text(settings_options[i], state.font, 15);
			option_text.setPosition(5, y_cord);
			y_cord += 18.0;
			option_texts[i] = option_text;
			float cur_text_width = option_text.getGlobalBounds().width;
			text_width = text_width > cur_text_width ? text_width : cur_text_width;
		}
		sf::RectangleShape settings_menu = sf::RectangleShape(sf::Vector2f(text_width + 10, text_y_size * 18 + 5));
		settings_menu.setFillColor(sf::Color(128, 128, 128, 128));
		settings_menu.setPosition(0, start_y_cord);

		state.window.draw(settings_menu);

		for (int i = 0; i < text_y_size; i++) {
			state.window.draw(option_texts[i]);
		}
	}
	state.window.draw(state.settings_button);
	state.window.draw(state.settings_button_text);
}
void gameOfLifeMain()
{
	printf("started\n");

	GOL_params state;
	printf("sizeof state:%d\n", sizeof(GOL_params));

	setup(state);
	board_reset_and_draw_pattern(state, 0);
	state.mainTexture.create(state.window_w, state.window_h);

	while (state.window.isOpen())
	{
		sf::Event event;
		while (state.window.pollEvent(event))
		{
			switch (event.type)
			{
			case sf::Event::Closed:
				state.window.close();
				break;
			case sf::Event::Resized:
				handle_window_resize(state, event.size.width, event.size.height, false);
				break;
			case sf::Event::MouseButtonPressed:
				handle_mouse_press(state, event.mouseButton.button);
				break;
			case sf::Event::MouseButtonReleased:
				handle_mouse_release(state, event.mouseButton.button);
				break;
			case sf::Event::MouseMoved:
				handle_mouse_move(state);
				break;
			case sf::Event::MouseWheelScrolled:
				handle_mouse_scroll(state, event.mouseWheelScroll.delta);
				break;
			case sf::Event::KeyReleased:
				handle_key_release(state, event.key.code);
				break;
			}
		}

		state.cur_time = clock();
		double ms_passed = (double)(state.cur_time - state.last_frame_time);
		double target_frame_time_ms = 1000.0 / state.simulation_fps;

		bool paused = !state.paused && state.drawMode == -1;
		bool frame_time_passed = ms_passed >= target_frame_time_ms;
		bool get_next_frame_and_draw = frame_time_passed && paused;

		bool redrawing = get_next_frame_and_draw || state.upload_and_redraw_board || state.redraw_board;

		if (redrawing) {
			hipError_t cudaStatus;
			if (get_next_frame_and_draw && !state.upload_and_redraw_board) {
				state.last_frame_time = state.cur_time;
				cudaStatus = get_next_frame_cuda(state);

				clock_t time_after_drawing_frame = clock();
				double frame_time = (double)(time_after_drawing_frame - state.last_frame_time);
				printf("frame time: %lf / %lf\n", frame_time, ms_passed);
			}

			if (state.upload_and_redraw_board && !state.MMousePressed) {
				cudaStatus = hipMemcpy(state.dev_board_IN, state.board, state.BOARD_AREA, hipMemcpyHostToDevice);
				state.upload_and_redraw_board = false;
			}

			cudaStatus = update_pixels_cuda(state);

			state.mainTexture.update(state.pixels, state.window_w, state.window_h, 0, 0);
			state.redraw_board = false;
		}

		state.mainSprite.setTexture(state.mainTexture, state.resetRect);
		state.resetRect = false;

		state.window.clear(sf::Color::Green);
		state.window.draw(state.mainSprite);

		draw_settings_sprite(state);

		state.window.display();
	}
}